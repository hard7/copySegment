#include "hip/hip_runtime.h"

#include "cuda_types.h"


__device__ long cudaTreadIndex() {}

extern "C"
__global__ void copySegment(char* srcPtr, cudaType::Point srcWidth, cudaType::Box srcSegment,
                            char* dstPtr, cudaType::Point dstWidth, cudaType::Box dstSegment, int sizeofType) {

    long tid = cudaTreadIndex();
    int xS = srcSegment.lo.x + (tid % srcSegment.width.x);
    int yS = srcSegment.lo.y + (tid / srcSegment.width.x % srcSegment.width.y);
    int zS = srcSegment.lo.z + (tid / srcSegment.width.x / srcSegment.width.y % srcSegment.width.z);

    int xD = dstSegment.lo.x + (tid % dstSegment.width.x);
    int yD = dstSegment.lo.y + (tid / dstSegment.width.x % dstSegment.width.y);
    int zD = dstSegment.lo.z + (tid / dstSegment.width.x / dstSegment.width.y % dstSegment.width.z);

    long offsetSrc = sizeofType * ( (long) srcWidth.y * srcWidth.x * zS + srcWidth.x * yS + xS);
    long offsetDst = sizeofType * ( (long) dstWidth.y * dstWidth.x * zD + dstWidth.x * yD + xD);


}




extern "C" __global__ void setup() {}